#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */


#include "ReduceSum.hpp"
#include <hip/hip_fp16.h>
#include <cassert>

nvinfer1::Dims ReduceSumPlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputDims,
                                                int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims output_dims = inputDims[0];
  output_dims.d[_axis] = 1;
  return output_dims;
}

int ReduceSumPlugin::initialize() {
  _output_dims = this->getOutputDimensions(0, &this->getInputDims(0), 1);
  return 0;
}

template <typename Data>
__global__ void reduce_sum(const int n,
      const int batchsize, const int channels,
      const int height, const int width,
      const Data *idata, Data* odata) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    for (int b = 0; b < batchsize; ++b) {
      Data sum = 0.;
      auto odx = idx - b * (channels - 1) * n;
      for (int c = 0; c < channels; ++c) {
        sum += idata[idx];
        idx += n;
      }
      odata[odx] = sum;
    }
  }
}

int ReduceSumPlugin::enqueue(int batchSize,
                                  const void *const *inputs, void **outputs,
                                  void *workspace, hipStream_t stream) {
  auto const& input_dims = this->getInputDims(0);
  const int channels = input_dims.d[0];
  const int input_height = input_dims.d[1];
  const int input_width = input_dims.d[2];
  const int num_kernels = input_height * input_width;
  const int num_threads = 512;
  const int num_blocks = (num_kernels + num_threads - 1) / num_threads;
  if (getDataType() == nvinfer1::DataType::kFLOAT) {
    reduce_sum<<<num_blocks, num_threads>>>(num_kernels, batchSize, channels,
      input_height, input_width,
      static_cast<float const*>(inputs[0]),
      static_cast<float*>(outputs[0]));
  } else {
    return -1;
  }
  return hipGetLastError() != hipSuccess;
}
