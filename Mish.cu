#include "hip/hip_runtime.h"
//
// Created by cao on 19-12-20.
//

#include "Mish.hpp"
#include <iostream>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <>

#define CHECK_CUDA(call) do {    \
  hipError_t status = call; \
  if( status != hipSuccess ) { \
    return status; \
  } \
} while(0)


template <typename T>
__device__ T softplus_kernel(T x, const T threshold) {
    if (x > threshold) return x;                // too large
    else if (x < -threshold) return expf(x);    // too small
    return logf(expf(x) + 1.);
}


// __device__ __half tanh_activate_kernel(__half x){return (__half(2.)/(__half(1.) + hexp(__half(-2.)*x)) - __half(1.));}

__device__ float tanh_activate_kernel(float x){return (2./(1. + expf(-2.*x)) - 1.);}

template <typename T>
__global__ void mishKernel( int n, const T* input, T* output, const T MISH_THRESHOLD)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        T x_val = input[idx];
        output[idx] = x_val * tanh_activate_kernel( softplus_kernel(x_val, MISH_THRESHOLD) );
    }
}

template <typename T>
inline int computeMish(hipStream_t stream, int n, const T* input, T* output)
{

    constexpr int blockSize = 1024;
    const int gridSize = (n + blockSize - 1) / blockSize;
    const T thr = 20.0;
    mishKernel<T><<<gridSize, blockSize, 0, stream>>>(n, input, output, thr);
    CHECK_CUDA(hipPeekAtLastError());
    return 0;
}

MishPlugin::MishPlugin():_initialized(false){


}
int MishPlugin::initialize() {
    if(_initialized) return 0;
    _initialized = true;
    return 0;
}
void MishPlugin::terminate() {
    if (!_initialized) {
        return;
    }
    _initialized = false;
}

MishPlugin::~MishPlugin() {
    terminate();
}

nvinfer1::Dims MishPlugin::getOutputDimensions(int index, const nvinfer1::Dims *inputDims, int nbInputs) {
    assert(index == 0);
    assert(inputDims);
    assert(nbInputs == 1);
    return inputDims[0];
}
size_t MishPlugin::getWorkspaceSize(int maxBatchSize) const {
    return 0;
}

int MishPlugin::enqueue(int batchSize, const void *const *inputs, void **outputs, void *workspace,
                         hipStream_t stream) {
    nvinfer1::Dims input_dims = this->getInputDims(0);
    nvinfer1::DataType type = this->getDataType();
    const int C = input_dims.d[0];
    const int H = input_dims.d[1];
    const int W = input_dims.d[2];
    const int num = batchSize*C*H*W;
    switch (type)
    {
        case nvinfer1::DataType::kFLOAT:
        {
            const float* input_data = static_cast<const float*>(inputs[0]);
            float* out_data= static_cast<float*>(outputs[0]);
            computeMish(stream,num,input_data,out_data);
            break;
        }
        case nvinfer1::DataType::kHALF:
        {
            return - 1;
        }
        default: std::cerr << "error data type" << std::endl;;
    }
    return 0;
}
