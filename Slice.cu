#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "Slice.hpp"
#include <hip/hip_fp16.h>
#include <cassert>

nvinfer1::Dims SlicePlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputDims,
                                                int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  assert(_axis == 0);
  nvinfer1::Dims const& input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  output_dims.d[_axis] = _end - _start;
  return output_dims;
}

int SlicePlugin::initialize() {
  _output_dims = this->getOutputDimensions(0, &this->getInputDims(0), 1);
  return 0;
}

template <typename Data>
__global__ void slice_channels(const int n,
        const int batchsize, const int channels,
        const int height, const int width,
        const int start, const int end,
        const Data* idata, Data* odata) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
      for (int b = 0; b < batchsize; b++) {
        for (int c = 0; c < channels; ++c) {
            if( c >= start && c < end){
              odata[idx - start * n - b * (channels + start - end) * n] = idata[idx];
            }
            idx += n;
        }
      }
    }
  }

int SlicePlugin::enqueue(int batchSize,
                                 const void *const *inputs, void **outputs,
                                 void *workspace, hipStream_t stream) {
  auto const& input_dims = this->getInputDims(0);
  const int channels = input_dims.d[0];
  // cout << "SlicePlugin batchSize: " << batchSize;
  // cout << " input_dims: " << input_dims.d[0] << " " << input_dims.d[1] << " " << input_dims.d[2];
  // cout << " output_dims: " << _output_dims.d[0] << " " << _output_dims.d[1] << " " << _output_dims.d[2] << endl;
  const int input_height = input_dims.d[1];
  const int input_width = input_dims.d[2];
  const int num_kernels = input_height * input_width;
  const int num_threads = 512;
  const int num_blocks = (num_kernels + num_threads - 1) / num_threads;
  if (getDataType() == nvinfer1::DataType::kFLOAT) {
    slice_channels<<<num_blocks, num_threads>>>(num_kernels, batchSize, channels, 
      input_height, input_width, _start, _end,
      static_cast<float const*>(inputs[0]),
      static_cast<float*>(outputs[0]));
  } else {
    slice_channels<<<num_blocks, num_threads>>>(num_kernels, batchSize, channels, 
      input_height, input_width, _start, _end,
      static_cast<half const*>(inputs[0]),
      static_cast<half*>(outputs[0]));
  }
  return hipGetLastError() != hipSuccess;
}
